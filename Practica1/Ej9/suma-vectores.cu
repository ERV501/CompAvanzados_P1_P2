#include "hip/hip_runtime.h"
#include <stdio.h>

#define N 500000
#define THREADS_N 32

__global__ void VecAdd(int* DA, int* DB, int* DC)
{
  int i=0;
  int stride = gridDim.x * blockDim.x;
  int start = (threadIdx.x + (blockIdx.x * blockDim.x)) * stride; // id_hebra + id_bloque x hebras_totales preparando el inicio del siguiente bloque en el final del anterior

  for(i=start; i < stride+start; i++){
    if(i <= N){
      DC[i] = DA[i] + DB[i];
    }
  }
}

int main()
{ 
  hipFree(0);
  
  int HA[N], HB[N], HC[N];
  int *DA, *DB, *DC;
  int i; 
  int size = N*sizeof(int);
  hipError_t aM,bM,cM,aN,bN,cN,e_kernel; //Guardar errores

  // reservamos espacio en la memoria global del device

  aM = hipMalloc((void**)&DA, size);
  printf(" hipMalloc DA: %s \n",hipGetErrorString(aM));

  bM = hipMalloc((void**)&DB, size);
  printf("hipMalloc DB: %s \n",hipGetErrorString(bM));

  cM = hipMalloc((void**)&DC, size);
  printf("hipMalloc DC: %s \n",hipGetErrorString(cM));
  
  // inicializamos HA y HB
  for (i=0; i<N; i++) {HA[i]=-i; HB[i] = 3*i;}
  
  // copiamos HA y HB del host a DA y DB en el device, respectivamente
  aN = hipMemcpy(DA, HA, size, hipMemcpyHostToDevice);
  printf(" hipMemcpy DA: %s \n",hipGetErrorString(aN));

  bN = hipMemcpy(DB, HB, size, hipMemcpyHostToDevice);
  printf(" hipMemcpy DB: %s \n",hipGetErrorString(bN));

  // llamamos al kernel (1 bloque de N hilos)
  
  dim3 dg, db; // tuplas de 3 dimensiones para grid y bloques
  
  
  dg.x = min(N/THREADS_N,1024);/*determinar bloques de 1 dimension*/
  db.x = THREADS_N;

  VecAdd <<<dg, db>>>(DA, DB, DC);
  e_kernel = hipGetLastError(); //Cojer ultimo error, ya que el kernel no devuelve ningun error_t
  printf(" kernel: %s \n",hipGetErrorString(e_kernel)); //Imprimir ultimo error

  // copiamos el resultado, que está en la memoria global del device, (DC) al host (a HC)
  cN = hipMemcpy(HC, DC, size, hipMemcpyDeviceToHost);
  printf(" hipMemcpy HC: %s \n",hipGetErrorString(cN));

  // liberamos la memoria reservada en el device
  hipFree(DA); hipFree(DB); hipFree(DC);  
  
  // una vez que tenemos los resultados en el host, comprobamos que son correctos
  // esta comprobación debe quitarse una vez que el programa es correcto (p. ej., para medir el tiempo de ejecución)
  for (i = 0; i < N; i++){

    if (HC[i]!= (HA[i]+HB[i])) 
		{       printf("pos:%d, %d + %d = %d\n",i,HA[i],HB[i],HC[i]);

      printf("error en componente %d\n", i); break;}
  }
  return 0;
} 
